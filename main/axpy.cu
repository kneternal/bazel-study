#include "hip/hip_runtime.h"
#include "lib/hello-time.h"
#include "main/hello-greet.h"
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
__global__ void axpy(float a, float* x, float* y) {
  y[threadIdx.x] = a * x[threadIdx.x];
}

int main(int argc, char** argv) {
  int devID = 0;
  hipDeviceProp_t deviceProp;
  hipError_t error = hipGetDevice(&devID);

  if (error != hipSuccess){
    printf("hipGetDevice returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
  }

  error = hipGetDeviceProperties(&deviceProp, devID);

  if (error != hipSuccess)
    {
      printf("hipGetDeviceProperties returned error %s (code %d), line(%d)\n", hipGetErrorString(error), error, __LINE__);
    }
  else
    {
      printf("GPU Device %d: \"%s\" with compute capability %d.%d\n\n", devID, deviceProp.name, deviceProp.major, deviceProp.minor);
    }
  
  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
  float host_y[kDataLen];

  // Copy input data to device.
  float* device_x;
  float* device_y;
  hipMalloc(&device_x, kDataLen * sizeof(float));
  hipMalloc(&device_y, kDataLen * sizeof(float));
  hipMemcpy(device_x, host_x, kDataLen * sizeof(float),
             hipMemcpyHostToDevice);

  // Launch the kernel.
  axpy<<<1, kDataLen>>>(a, device_x, device_y);

  // Copy output data to host.
  hipDeviceSynchronize();
  hipMemcpy(host_y, device_y, kDataLen * sizeof(float),
             hipMemcpyDeviceToHost);

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << host_y[i] << "\n";
  }

  hipDeviceReset();


  std::string who = "world";
  if (argc > 1) {
    who = argv[1];
  }
  std::cout << get_greet(who) << std::endl;
  print_localtime();
  return 0;
}
